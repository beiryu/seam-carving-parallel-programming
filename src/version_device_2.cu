#include "hip/hip_runtime.h"
#include "gpu2.cuh"
#include "utils.cuh"

void run_gpu2(char *image_path, Debuger *debuger, int nSeams)
{
    int blocksize = 32;

    char gpu_savepath[] = "gpu_2.pnm";
    GpuTimer timer;
    GPU2 *device = new GPU2();

    // int list_nseams[5] = {300, 50, 100, 150, 300};
    int width, height;
    uchar3 *inPixels = NULL;
    uchar3 *outPixels = NULL;

    readPnm(image_path, width, height, inPixels);
    for (int i = 0; i < 1; i++)
    {

        timer.Start();
        device->applySeamCarving(inPixels, width, height, nSeams, outPixels, blocksize, debuger);
        timer.Stop();

        float kernelTime = timer.Elapsed();
        printf("Version GPU 2, %d seams: %f ms\n", nSeams, kernelTime);

        writePnm(outPixels, width - nSeams, height, gpu_savepath);
    }
}

int main(int argc, char **argv)
{
    char debug_folder[] = "debug_device2/";
    char seams_folder[] = "seams/";
    char energy_filename[] = "energy.pnm";
    Debuger *debuger = new Debuger(debug_folder, energy_filename, seams_folder, 10, false);

    run_gpu2(argv[1], debuger, atoi(argv[2]));
    return 0;
}