#include "hip/hip_runtime.h"
#include "gpu1.cuh"
#include "utils.cuh"

void run_gpu1(char *image_path, Debuger *debuger, int nSeams)
{
    int blocksize = 32;

    char gpu_savepath[] = "gpu_1.pnm";
    GpuTimer timer;
    GPU1 *device = new GPU1();

    // int list_nseams[5] = {300, 50, 100, 150, 300};
    int width, height;
    uchar3 *inPixels = NULL;
    uchar3 *outPixels = NULL;

    readPnm(image_path, width, height, inPixels);
    for (int i = 0; i < 1; i++)
    {

        timer.Start();
        device->applySeamCarving(inPixels, width, height, nSeams, outPixels, blocksize, debuger);
        timer.Stop();

        float kernelTime = timer.Elapsed();
        printf("Version GPU 1, %d seams: %f ms\n", nSeams, kernelTime);

        writePnm(outPixels, width - nSeams, height, gpu_savepath);
    }
}
int main(int argc, char **argv)
{
    char debug_folder[] = "debug_device1/";
    char seams_folder[] = "seams/";
    char energy_filename[] = "energy.pnm";
    Debuger *debuger = new Debuger(debug_folder, energy_filename, seams_folder, 10, false);

    run_gpu1(argv[1], debuger, atoi(argv[2]));
    return 0;
}