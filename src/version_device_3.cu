#include "hip/hip_runtime.h"
#include "gpu3.cuh"
#include "utils.cuh"

void run_gpu3(char *image_path, Debuger *debuger, int nSeams)
{
    int blocksize = 32;

    char gpu_savepath[] = "gpu_3.pnm";
    GpuTimer timer;
    GPU3 *device = new GPU3();

    int width, height;
    uchar3 *inPixels = NULL;
    uchar3 *outPixels = NULL;

    readPnm(image_path, width, height, inPixels);
    for (int i = 0; i < 1; i++)
    {

        timer.Start();
        device->applySeamCarving(inPixels, width, height, nSeams, outPixels, blocksize, debuger);
        timer.Stop();

        float kernelTime = timer.Elapsed();
        printf("Version GPU 3, %d seams: %f ms\n", nSeams, kernelTime);

        writePnm(outPixels, width - nSeams, height, gpu_savepath);
    }
}
int main(int argc, char **argv)
{
    char debug_folder[] = "debug_device3/";
    char seams_folder[] = "seams/";
    char energy_filename[] = "energy.pnm";
    Debuger *debuger = new Debuger(debug_folder, energy_filename, seams_folder, 10, false);

    run_gpu3(argv[1], debuger, atoi(argv[2]));
    return 0;
}